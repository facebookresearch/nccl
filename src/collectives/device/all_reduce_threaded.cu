#include "hip/hip_runtime.h"
// (c) Meta Platforms, Inc. and affiliates. Confidential and proprietary.

#include "all_reduce.h"
#include "all_reduce_threaded.h"
#include "common.h"
#include "collectives.h"

#define idx(nranks, i, j) ((i) * (nranks) + (j))

template <typename T>
static inline __device__ uint32_t vecElementAdd(const uint32_t& a, const uint32_t& b)
{
    if (std::is_same<T, half>::value) {
        const __half *x = reinterpret_cast<const __half*>(&a);
        const __half *y = reinterpret_cast<const __half*>(&b);

        __half2 p = __halves2half2(x[0], x[1]);
        __half2 q = __halves2half2(y[0], y[1]);

        __half2 z = __hadd2(p, q);
        return (reinterpret_cast<uint32_t *>(&z))[0];
#if defined(__CUDA_BF16_TYPES_EXIST__)
    } else if (std::is_same<T, __hip_bfloat16>::value) {
        const __hip_bfloat16 *x = reinterpret_cast<const __hip_bfloat16*>(&a);
        const __hip_bfloat16 *y = reinterpret_cast<const __hip_bfloat16*>(&b);

#if (__CUDA_ARCH__ >= 800)
        __hip_bfloat162 p = { x[0], x[1] };
        __hip_bfloat162 q = { y[0], y[1] };

        __hip_bfloat162 z = __hadd2(p, q);
        return (reinterpret_cast<uint32_t *>(&z))[0];
#else
        __hip_bfloat16 z[2] = { x[0] + y[0], x[1] + y[1] };
        return (reinterpret_cast<uint32_t *>(z))[0];
#endif
    }
#endif

    return 0;
}

/* create a special version of seqAdd that can be disabled at
 * compile-time for bfloat16 (using enable_if).  This is because the
 * base version of seqAdd does not compile for bfloat16, so we are
 * essentially tricking the compiler.  We never call this version for
 * bfloat16, so it doesn't matter that it does not compile, but the
 * compiler unfortunately does not know that. */
#if defined(__CUDA_BF16_TYPES_EXIST__)
template <typename T, uint32_t NRANKS>
static inline __device__
typename std::enable_if<!std::is_same<T,__hip_bfloat16>::value, uint4>::type
seqAdd(const T **src, size_t offset)
{
    T dst[16 / sizeof(T)] = { 0 };
    for (int i = 0; i < NRANKS; i++) {
        uint4 vals = reinterpret_cast<const uint4 *>(&src[i][offset])[0];
        const T *src_d = reinterpret_cast<const T *>(&vals);
        for (int j = 0; j < 16 / sizeof(T); j++) {
            dst[j] += src_d[j];
        }
    }
    return reinterpret_cast<uint4 *>(&dst)[0];
}

template <typename T, uint32_t NRANKS>
static inline __device__
typename std::enable_if<std::is_same<T,__hip_bfloat16>::value, uint4>::type
seqAdd(const T **src, size_t offset)
{
    uint4 x = { 0, 0, 0, 0 };

    return x;
}

#else

template <typename T, uint32_t NRANKS>
static inline __device__ uint4 seqAdd(const T **src, size_t offset)
{
    T dst[16 / sizeof(T)] = { 0 };
    for (int i = 0; i < NRANKS; i++) {
        /* 16-byte load */
        uint4 vals = reinterpret_cast<const uint4 *>(&src[i][offset])[0];

        /* sequential additions */
        const T *src_d = reinterpret_cast<const T *>(&vals);
        for (int j = 0; j < 16 / sizeof(T); j++) {
            dst[j] += src_d[j];
        }
    }
    return reinterpret_cast<uint4 *>(&dst)[0];
}

#endif

template <typename T, uint32_t NRANKS>
static inline __device__ uint4 vecAdd(const T **src, size_t offset)
{
    if (std::is_same<T, half>::value
#if defined(__CUDA_BF16_TYPES_EXIST__)
        || std::is_same<T, __hip_bfloat16>::value
#endif
    ) {
        uint4 dst = { 0, 0, 0, 0 };
        for (int i = 0; i < NRANKS; i++) {
            /* 16-byte load */
            uint4 vals = reinterpret_cast<const uint4 *>(&src[i][offset])[0];

            /* vector additions */
            dst.x = vecElementAdd<T>(dst.x, vals.x);
            dst.y = vecElementAdd<T>(dst.y, vals.y);
            dst.z = vecElementAdd<T>(dst.z, vals.z);
            dst.w = vecElementAdd<T>(dst.w, vals.w);
        }
        return dst;
    } else {
        return seqAdd<T,NRANKS>(src, offset);
    }
}

template <typename T>
static inline __device__ uint4 vecAdd(const T *src_a, const T *src_b)
{
    /* 16-byte loads */
    uint4 vals_a = reinterpret_cast<const uint4 *>(src_a)[0];
    uint4 vals_b = reinterpret_cast<const uint4 *>(src_b)[0];

    if (std::is_same<T, half>::value
#if defined(__CUDA_BF16_TYPES_EXIST__)
        || std::is_same<T, __hip_bfloat16>::value
#endif
    ) {
        /* vector additions */
        uint4 dst;
        dst.x = vecElementAdd<T>(vals_a.x, vals_b.x);
        dst.y = vecElementAdd<T>(vals_a.y, vals_b.y);
        dst.z = vecElementAdd<T>(vals_a.z, vals_b.z);
        dst.w = vecElementAdd<T>(vals_a.w, vals_b.w);
        return dst;
    } else {
        /* cast back to original type and do sequential additions */
        T dst[16 / sizeof(T)];
        const T *src_a_loaded = reinterpret_cast<const T *>(&vals_a);
        const T *src_b_loaded = reinterpret_cast<const T *>(&vals_b);
        for (int j = 0; j < 16 / sizeof(T); j++) {
            dst[j] = src_a_loaded[j] + src_b_loaded[j];
        }
        return reinterpret_cast<uint4 *>(&dst)[0];
    }
}

/*
 * Barrier Algorithm --
 * Consider the barrier mailbox as a 2D array (numranks x numranks).
 * A group of threads in each rank (global thread ID < numranks), set
 * their local source buffer address in the column corresponding to
 * their rank.  Then a group of threads in each block (block local
 * thread ID < numranks), check to see if the row corresponding to
 * their rank is set.  Finally, all threads in the block synchronize.
 *
 * Memory consistency --
 * Because the barrier mailbox is volatile, we do not need to worry
 * about register caching and the barrier data will always be
 * propagated through cache consistency.  Typically, we would store
 * the source buffer address in a buffer and then perform a barrier
 * synchronization to let the remaining ranks know that the source
 * buffer is ready.  However, because the GPU only maintains weak
 * ordering of store operations, the compiler or hardware could
 * reorder the store of the source buffer from the barrier store,
 * which can lead to incorrect results.  We would need to separate
 * these operations with a __threadfence_system() call to ensure store
 * ordering, which is expensive.  We workaround that by merging the
 * source buffer distribution with the barrier operation by using a
 * bit OR'ed combination of the source buffer address and the barrier
 * flag simultaneously.  Per the CUDA memory consistency semantics
 * defined in
 * https://docs.nvidia.com/cuda/cuda-c-programming-guide/index.html#memory-fence-functions,
 * this should ensure that the barrier flag and the source buffer
 * should both be visible to the other ranks simultaneously.
 *
 * Mailbox reuse across collective operations --
 * The barrier mailbox might be reused across collective operations,
 * so we need to ensure that we are not reading the buffer address
 * from the previous iteration.  Because our algorithm is only
 * applicable for 16-byte aligned addresses, the last four bits of the
 * buffer address are always zero.  We use the last bit of the address
 * buffer to indicate whether the data is valid or not, by swapping
 * between 1 and 0 between each iteration.  To retrieve the actual
 * source buffer address, we simply mask that last bit.
 */
template <uint32_t NRANKS>
static inline __device__ void barrier(uintptr_t *barrierPtrs, uintptr_t barrierFlag, int rank)
{
    volatile uintptr_t *barrier_d = barrierPtrs;

    if (threadIdx.x < NRANKS) {
        /* first block sets barrier values */
        if (blockIdx.x == 0) {
            barrier_d[idx(NRANKS, threadIdx.x, rank)] = barrierFlag;
        }

        /* all blocks check for values to be set */
        while ((barrier_d[idx(NRANKS, rank, threadIdx.x)] & 1UL) != (barrierFlag & 1UL)) {}
    }

    /* remaining threads in the block wait */
    __syncthreads();
}

/* We use a simple Allgather + local reduce algorithm here.  For small
 * messages, we are mostly latency bound on fast networks such as
 * NVLink.  So fetching data from all the GPUs simultaneously should
 * basically take the same amount of time as fetching data from one
 * GPU.  This algorithm directly reads data from the other GPUs and
 * reduces it into the local destination buffer. */
template <typename T, uint32_t NRANKS>
static inline __device__ void allreduceFlat(uintptr_t *barrierMbox, uintptr_t barrierFlag, int rank,
                                            const T *sendbuff, T *recvbuff, size_t count)
{
    const int gtidx = threadIdx.x + blockDim.x * blockIdx.x;

    /* global barrier */
    barrier<NRANKS>(barrierMbox, (reinterpret_cast<uintptr_t>(sendbuff)) | barrierFlag, rank);

    /* fetch remote source buffer addresses from the barrier mailbox */
    const T *src[NRANKS];
    for (int i = 0; i < NRANKS; i++) {
        src[i] = reinterpret_cast<const T *>
            (barrierMbox[idx(NRANKS, rank, (rank + i) & (NRANKS - 1))] & ~1UL);
    }

    for (size_t offset = gtidx * 16 / sizeof(T); offset < count; offset += gridDim.x * blockDim.x * 16 / sizeof(T)) {
        reinterpret_cast<uint4 *>(&recvbuff[offset])[0] = vecAdd<T,NRANKS>(src, offset);
    }
}

/* Hierarchical algorithm for slightly larger (but still
 * latency-sensitive) messages.  In this algorithm, we avoid every
 * rank fetching all of the data from every other rank that the flat
 * algorithm above does.  Instead, each rank fetches only a subset of
 * data from all other ranks and reduces locally.  Then we do a second
 * step where the reduced data is Allgathered (by direct copy by each
 * rank). */
template <typename T, uint32_t NRANKS>
static inline __device__ void allreduceTree(uintptr_t *barrierMbox, uintptr_t barrierFlag, int rank,
                                             const T *sendbuff, T *tmpbuff, T *recvbuff, size_t count)
{
    const int gtidx = threadIdx.x + blockDim.x * blockIdx.x;

    /* global barrier */
    barrier<NRANKS>(barrierMbox, (reinterpret_cast<uintptr_t>(sendbuff)) | barrierFlag, rank);

    const T *src[NRANKS];
    for (int i = 0; i < NRANKS; i++) {
        int r = (rank + i) & (NRANKS - 1);
        src[i] = reinterpret_cast<const T *> (barrierMbox[idx(NRANKS, rank, r)] & ~1UL);
    }

    size_t offsetStart = gtidx * 16 / sizeof(T);
    size_t offsetMax = count / NRANKS;
    size_t offsetStride = NRANKS * gridDim.x * blockDim.x * 16 / sizeof(T);

    for (size_t offset = offsetStart; offset < offsetMax; offset += offsetStride) {
        reinterpret_cast<uint4 *>(&recvbuff[offset])[0] = vecAdd<T,NRANKS>(src, offset + rank * count / NRANKS);
    }

    /* we cannot avoid a __threadfence_system() here because the next
     * step requires us to access the data that just got reduced by
     * the other ranks.  So we need to tell the compiler/hardware to
     * not reorder the above reduction to happen after the below
     * Allgather. */
    __threadfence_system();

    /* global barrier */
    barrier<NRANKS>(barrierMbox + NRANKS * NRANKS,
                    (reinterpret_cast<uintptr_t>(tmpbuff)) | barrierFlag, rank);

    int rankOffset[NRANKS];
    for (int i = 0; i < NRANKS; i++) {
        int r = (rank + i) & (NRANKS - 1);
        src[i] = reinterpret_cast<const T *> (barrierMbox[NRANKS * NRANKS + idx(NRANKS, rank, r)] & ~1UL);
        rankOffset[i] = r * count / NRANKS;
    }

    /* simple direct-access Allgather in 16-byte loads */
    for (size_t offset = offsetStart; offset < offsetMax; offset += offsetStride) {
        for (int i = 0; i < NRANKS; i++) {
            reinterpret_cast<uint4 *>(&recvbuff[offset + rankOffset[i]])[0] =
                reinterpret_cast<const uint4 *>(&src[i][offset])[0];
        }
    }
}

template <typename T, uint32_t NRANKS>
static inline __device__ void peerReduce(uintptr_t *localMbox, uintptr_t *peerMbox, T *tmpbuff, T *recvbuff,
                                         size_t count)
{
    volatile uintptr_t *peerMboxV = peerMbox;
    volatile uintptr_t *localMboxV = localMbox;

    if (threadIdx.x == 0) {
        if (blockIdx.x == 0) {
            *peerMboxV = reinterpret_cast<uintptr_t>(tmpbuff);
        }
        while (*localMboxV == 0) {}
    }
    __syncthreads();

    const T *src = reinterpret_cast<const T *>(*localMboxV);
    const int gtidx = threadIdx.x + blockDim.x * blockIdx.x;

    /* simple reduction with one peer rank */
    for (size_t offset = gtidx * 16 / sizeof(T); offset < count; offset += gridDim.x * blockDim.x * 16 / sizeof(T)) {
        reinterpret_cast<uint4 *>(&recvbuff[offset])[0] = vecAdd<T>(&src[offset], (const T *) &tmpbuff[offset]);
    }

    *localMbox = 0;
}

template <typename T, uint32_t NRANKS>
__global__ void ncclKernel_AllReduce_Threaded_Flat(uintptr_t *barrierMbox,
                                                   uintptr_t barrierFlag, int rank,
                                                   const T *sendbuff, T *recvbuff, size_t count)
{
    allreduceFlat<T,NRANKS>(barrierMbox, barrierFlag, rank, sendbuff, recvbuff, count);
}

template <typename T, uint32_t NRANKS>
__global__ void ncclKernel_AllReduce_Threaded_Tree(uintptr_t *barrierMbox, uintptr_t barrierFlag, int rank,
                                                   const T *sendbuff, T *tmpbuff, T *recvbuff, size_t count)
{
    allreduceTree<T,NRANKS>(barrierMbox, barrierFlag, rank, sendbuff, tmpbuff, recvbuff, count);
}

template <typename T, uint32_t NRANKS>
__global__ void ncclKernel_AllReduce_Threaded_HCM_Flat(uintptr_t *cliqueBarrierMbox, uintptr_t *localMbox,
                                                       uintptr_t *peerMbox, uintptr_t barrierFlag, int cliqueRank,
                                                       const T *sendbuff, T *tmpbuff, T *recvbuff, size_t count)
{
    /* For HCM systems, we break the Allreduce into two parts.  In the
     * first part, we perform the Allreduce within the clique (the set
     * of ranks that are topologically all-to-all connected with
     * direct NVLink connections), i.e., within the "mesh".  Then each
     * rank reduces data with its peer on the other mesh, i.e., across
     * the "cube".  We only support two meshes currently: this should
     * be sufficient for ZionEx and Zion4S.  It is unclear if NVIDIA
     * has other platforms that have a more generalized version of
     * HCM, so this code does not support the fully general case of
     * multidimensional cubes. */
    allreduceFlat<T,NRANKS/2>(cliqueBarrierMbox, barrierFlag, cliqueRank, sendbuff, tmpbuff, count);
    __threadfence_system();
    peerReduce<T,NRANKS>(localMbox, peerMbox, tmpbuff, recvbuff, count);
}

template <typename T, uint32_t NRANKS>
__global__ void ncclKernel_AllReduce_Threaded_HCM_Tree(uintptr_t *cliqueBarrierMbox, uintptr_t *localMbox,
                                                       uintptr_t *peerMbox, uintptr_t barrierFlag, int cliqueRank,
                                                       const T *sendbuff, T *tmpbuff, T *recvbuff, size_t count)
{
    /* using the recvbuff as a temporary buffer, so the output of
     * allreduce_tree goes into tmpbuff */
    allreduceTree<T,NRANKS/2>(cliqueBarrierMbox, barrierFlag, cliqueRank, sendbuff, recvbuff, tmpbuff, count);
    __threadfence_system();
    peerReduce<T,NRANKS>(localMbox, peerMbox, tmpbuff, recvbuff, count);
}

DECL_THREADED_FUNC(char);
DECL_THREADED_FUNC(uint8_t);
DECL_THREADED_FUNC(int32_t);
DECL_THREADED_FUNC(uint32_t);
DECL_THREADED_FUNC(int64_t);
DECL_THREADED_FUNC(uint64_t);
DECL_THREADED_FUNC(half);
DECL_THREADED_FUNC(float);
DECL_THREADED_FUNC(double);
#if defined(__CUDA_BF16_TYPES_EXIST__)
DECL_THREADED_FUNC(__hip_bfloat16);
#endif
