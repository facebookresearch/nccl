#include "hip/hip_runtime.h"

// (c) Meta Platforms, Inc. and affiliates. Confidential and proprietary.

#include <gmock/gmock.h>
#include <gtest/gtest.h>
#include <nccl.h>
#include <iostream>
#include "CtranAlgoDev.h"
#include "CtranGpe.h"
#include "CtranGpeDev.h"
#include "CtranGpeImpl.h"
#include "CtranGpeKernel.h"
#include "checks.h"
#include "nccl_cvars.h"
#include "tests_common.cuh"

class CtranGpeTest : public ::testing::Test {
 public:
  CtranGpe* gpe;
  int cudaDev;
  ncclComm_t dummyComm;
  CtranAlgoDeviceState* dummyDevState_d {nullptr};

  CtranGpeTest() = default;

 protected:
  void SetUp() override {
    cudaDev = 0;
    gpe = nullptr;

    CUDACHECK_TEST(hipMalloc(&dummyDevState_d, sizeof(CtranAlgoDeviceState)));
  }
  void TearDown() override {
    if (gpe != nullptr) {
      delete gpe;
    }
    CUDACHECK_TEST(hipFree(dummyDevState_d));
  }
};

class CtranGpeKernelTest : public ::testing::Test {
 public:
  volatile int* testFlag;
  CtranAlgoDeviceState* dummyDevState_d {nullptr};
  int cudaDev;
  CtranGpeKernelTest() = default;

 protected:
  void SetUp() override {
    cudaDev = 0;
    CUDACHECKIGNORE(hipSetDevice(cudaDev));
    CUDACHECKIGNORE(
        hipHostAlloc((void**)&testFlag, sizeof(int), hipHostMallocDefault));
    *testFlag = UNSET;

    CUDACHECK_TEST(hipMalloc(&dummyDevState_d, sizeof(CtranAlgoDeviceState)));
  }
  void TearDown() override {
    CUDACHECKIGNORE(hipHostFree((void*)testFlag));
    CUDACHECK_TEST(hipFree(dummyDevState_d));
  }
};

constexpr std::string_view kExpectedOutput = "CtranGpeTestAlgoFunc Called";
static ncclResult_t CtranGpeTestAlgoFunc(
    std::vector<std::unique_ptr<struct OpElem>> opGroup) {
  std::cout << kExpectedOutput;
  return ncclSuccess;
}

__global__ void CtranGpeTestKernel(
    int* flag,
    CtranAlgoDeviceState* devState_d,
    CtranKernelAllGatherArgs args) {
  int* a = const_cast<int*>(reinterpret_cast<const int*>(args.sendbuff));
  int expValInt = reinterpret_cast<int>(args.recvbuff);
  size_t count = args.nbytes;

  if (flag) {
    ncclKernelStartGpe(flag);
  }

  for (int i = 0; i < count; i++) {
    a[i] = expValInt;
  }

  if (flag) {
    ncclKernelWaitGpeTerminate(flag);
  }
}

__global__ void CtranGpeTestTerminateKernel(int* flag) {
  ncclKernelStartGpe(flag);
  ncclKernelWaitGpeTerminate(flag);
}

constexpr int numP2pElems = 10;
__global__ void CtranGpeTestP2pElemsKernel(
    int* flag,
    CtranAlgoDeviceState* devState_d,
    CtranKernelAllGatherArgs args) {
  KernelP2pElem* elemList = const_cast<KernelP2pElem*>(
      reinterpret_cast<const KernelP2pElem*>(args.sendbuff));
  KernelP2pElem* elem = elemList;
  int numElems = numP2pElems;
  // consume only numP2pElems amount of objects
  while (numElems) {
    elem->inuse[blockIdx.x] = false;
    elem = elem->next;
    numElems--;
  }
}

TEST_F(CtranGpeTest, gpeThread) {
  gpe = new CtranGpe(cudaDev);
  EXPECT_THAT(gpe, testing::NotNull());
}

TEST_F(CtranGpeTest, SubmitOpBadArgs) {
  ncclResult_t res = ncclSuccess;
  CtranGpe* gpe = new CtranGpe(cudaDev);

  std::vector<std::unique_ptr<struct OpElem>> ops;
  struct OpElem* op;
  op = new struct OpElem(OpElem::opType::SEND, dummyComm);
  op->send.sendbuff = nullptr;
  op->send.count = 0;
  op->send.datatype = ncclInt8;
  op->send.peerRank = 0;
  ops.push_back(std::unique_ptr<struct OpElem>(op));

  auto kernelConfig = KernelConfig(KernelConfig::KernelType::SEND, nullptr);
  kernelConfig.args.devState_d = dummyDevState_d;

  /* NOTE: invalid CUDA kernel should return error code */
  res =
      gpe->submit(std::move(ops), &CtranGpeTestAlgoFunc, kernelConfig, nullptr);
  EXPECT_NE(res, ncclSuccess);

  // Invalid devState_d should be checked and return ncclInternalError
  kernelConfig.args.devState_d = nullptr;
  res = gpe->submit(
      std::move(ops),
      &CtranGpeTestAlgoFunc,
      kernelConfig,
      reinterpret_cast<void*>(CtranGpeTestKernel));
  EXPECT_EQ(res, ncclInternalError);
}

constexpr int count = 1024;
constexpr int kKernelpdatedVal = 100;

TEST_F(CtranGpeTest, SubmitOpKernel) {
  ncclResult_t res = ncclSuccess;
  CtranGpe* gpe = new CtranGpe(cudaDev);
  hipStream_t stream;
  CUDACHECK_TEST(hipStreamCreate(&stream));

  int* a = nullptr;
  CUDACHECK_TEST(hipMalloc(&a, sizeof(int) * count));
  CUDACHECK_TEST(hipMemset(a, 0, sizeof(int) * count));
  CUDACHECK_TEST(hipDeviceSynchronize());

  std::vector<std::unique_ptr<struct OpElem>> ops;
  struct OpElem* op;
  op = new struct OpElem(OpElem::opType::RECV, dummyComm);
  op->recv.recvbuff = nullptr;
  op->recv.count = 0;
  op->recv.datatype = ncclInt8;
  op->recv.peerRank = 0;
  ops.push_back(std::unique_ptr<struct OpElem>(op));

  // Use ALLGATHER kernel config to pass test variables
  auto config = KernelConfig(KernelConfig::KernelType::ALLGATHER, stream);
  ctranKernelSetAllGatherArgs(
      a,
      reinterpret_cast<void*>(kKernelpdatedVal),
      count,
      dummyDevState_d,
      &config.args);

  testing::internal::CaptureStdout();

  res = gpe->submit(
      std::move(ops),
      &CtranGpeTestAlgoFunc,
      config,
      reinterpret_cast<void*>(CtranGpeTestKernel));

  EXPECT_EQ(res, ncclSuccess);

  CUDACHECK_TEST(hipStreamDestroy(stream));
  delete gpe;
  gpe = nullptr;

  // check GPE hostFn has been called
  std::string output = testing::internal::GetCapturedStdout();
  EXPECT_THAT(output, testing::HasSubstr(kExpectedOutput));

  // check kernel has been called
  std::vector<int> a_host(count, 0);
  CUDACHECK_TEST(hipMemcpy(
      a_host.data(), a, sizeof(int) * count, hipMemcpyDeviceToHost));
  EXPECT_THAT(a_host, testing::Each(kKernelpdatedVal));
}

TEST_F(CtranGpeTest, SubmitOnlyKernel) {
  ncclResult_t res = ncclSuccess;
  auto gpe = std::unique_ptr<CtranGpe>(new CtranGpe(cudaDev));
  hipStream_t stream;
  CUDACHECK_TEST(hipStreamCreate(&stream));

  int* a = nullptr;
  CUDACHECK_TEST(hipMalloc(&a, sizeof(int) * count));
  CUDACHECK_TEST(hipMemset(a, 0, sizeof(int) * count));
  CUDACHECK_TEST(hipDeviceSynchronize());

  std::vector<std::unique_ptr<struct OpElem>> emptyOps;

  // Use ALLGATHER kernel config to pass test variables
  auto config = KernelConfig(KernelConfig::KernelType::ALLGATHER, stream);
  ctranKernelSetAllGatherArgs(
      a,
      reinterpret_cast<void*>(kKernelpdatedVal),
      count,
      dummyDevState_d,
      &config.args);

  // empty OpGroup would launch only kernel
  res = gpe->submit(
      std::move(emptyOps),
      nullptr,
      config,
      reinterpret_cast<void*>(CtranGpeTestKernel));
  EXPECT_EQ(res, ncclSuccess);
  CUDACHECK_TEST(hipStreamSynchronize(stream));

  // check kernel has been called
  std::vector<int> a_host(count, 0);
  CUDACHECK_TEST(hipMemcpy(
      a_host.data(), a, sizeof(int) * count, hipMemcpyDeviceToHost));
  EXPECT_THAT(a_host, testing::Each(kKernelpdatedVal));

  CUDACHECK_TEST(hipFree(a));
  CUDACHECK_TEST(hipStreamDestroy(stream));
}

TEST_F(CtranGpeKernelTest, launchTerminateStallKernel) {
  dim3 grid = {1, 1, 1};
  dim3 blocks = {1, 1, 1};
  void* args[] = {&testFlag};
  auto res = hipLaunchKernel(
      reinterpret_cast<void*>(CtranGpeTestTerminateKernel),
      grid,
      blocks,
      args,
      0,
      0);

  EXPECT_EQ(res, hipSuccess);

  while (*testFlag != KERNEL_STARTED) {
    EXPECT_THAT(*testFlag, testing::Not(KERNEL_TERMINATE));
  }

  *testFlag = KERNEL_TERMINATE;
  res = hipStreamSynchronize(0);

  EXPECT_EQ(res, hipSuccess);
}

TEST_F(CtranGpeKernelTest, SubmitKernelWithP2pElems) {
  // Ensure NCCL_CTRAN_NUM_KERNEL_P2PELEMS has been set
  ncclCvarInit();
  auto gpe = std::unique_ptr<CtranGpe>(new CtranGpe(cudaDev));
  hipStream_t stream;
  CUDACHECK_TEST(hipStreamCreate(&stream));

  // Allocate p2pElems
  KernelP2pElem* elemList = nullptr;
  constexpr int ngroups = 5;
  NCCLCHECK_TEST(gpe->allocKernelP2pElems(numP2pElems, ngroups, &elemList));

  // Check allocated number of p2pElems is as expected
  int nAllocated = 0;
  KernelP2pElem* elem = elemList;
  while (elem) {
    elem = elem->next;
    nAllocated++;
  }
  EXPECT_EQ(nAllocated, numP2pElems);

  // Use ALLGATHER kernel config to pass test variables and launch with ngroups
  // gridSize to consume the elems
  std::vector<std::unique_ptr<struct OpElem>> emptyOps;
  auto config = KernelConfig(KernelConfig::KernelType::ALLGATHER, stream);
  ctranKernelSetAllGatherArgs(elemList, nullptr, 0, dummyDevState_d, &config.args);
  config.numBlocks = ngroups;

  // Empty OpGroup would launch only kernel
  NCCLCHECK_TEST(gpe->submit(
      std::move(emptyOps),
      nullptr,
      config,
      reinterpret_cast<void*>(CtranGpeTestP2pElemsKernel)));

  CUDACHECK_TEST(hipStreamSynchronize(stream));

  // Check each element has been consumed by kernel
  elem = elemList;
  while (elem) {
    std::vector<int> inuse(elem->inuse, elem->inuse + ngroups);
    EXPECT_THAT(inuse, testing::Each(false));
    elem = elem->next;
  }

  // Skip check for reclaim which is an internal operation and triggered in GPE
  // destructor. Coverd by separate UT

  CUDACHECK_TEST(hipStreamDestroy(stream));
}
