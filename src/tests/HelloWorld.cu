#include <stdio.h>
#include "hip/hip_runtime.h"
#include "nccl.h"
#include "mpi.h"
#include <unistd.h>
#include <stdint.h>
#include <stdlib.h>
#include "tests_common.cuh"

int main(int argc, char* argv[])
{
  int size = 32*1024;

  int localRank, globalRank, numRanks;

  ncclComm_t comm;
  float *sendbuff, *recvbuff;
  hipStream_t s;

  std::tie(localRank, globalRank, numRanks, comm) = setupNccl(argc, argv);

  CUDACHECK_TEST(hipMalloc(&sendbuff, size * sizeof(float)));
  CUDACHECK_TEST(hipMalloc(&recvbuff, size * sizeof(float)));
  CUDACHECK_TEST(hipStreamCreate(&s));

  for (int x = 0; x < 32; x++) {
    //communicating using NCCL
    NCCLCHECK_TEST(ncclAllReduce((const void*)sendbuff, (void*)recvbuff, size, ncclFloat, ncclSum,
          comm, s));
  }

  for (int x = 0; x < 32; x++) {
    NCCLCHECK_TEST(ncclAllToAll(sendbuff, recvbuff, size/2, ncclFloat, comm, s));
  }

  //completing NCCL operation by synchronizing on the CUDA stream
  CUDACHECK_TEST(hipStreamSynchronize(s));

  //free device buffers
  CUDACHECK_TEST(hipFree(sendbuff));
  CUDACHECK_TEST(hipFree(recvbuff));

  cleanupNccl(comm);

  printf("[MPI Rank %d] Success \n", globalRank);
  return 0;
}
